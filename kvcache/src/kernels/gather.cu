#include "hip/hip_runtime.h"
#include "config.h"
#include "kernels/gather.h"

#include <stdexcept>

namespace atex::ops {

/**
 * @brief CUDA kernel for aligned gathering of cache data.
 *
 * @tparam TPB Threads per block.
 * @tparam VPT Number of elements processed per thread.
 */
template <int64_t TPB, int64_t VPT>
__global__ void device_cache_gather_align16(const int32_t *__restrict__ src, int32_t *__restrict__ dst,
                                            const int32_t *__restrict__ page_idx_array, const int64_t num_of_page,
                                            const int64_t num_of_layer, const int64_t page_size,
                                            const int64_t layer_stride, const int64_t page_stride) {

  for (int64_t page_idx = blockIdx.x; page_idx < num_of_page; page_idx += gridDim.x) {
    const int64_t src_page_offset = page_idx_array[page_idx] * page_stride;
    const int64_t dst_page_offset = page_idx * page_size;

    for (int64_t offset = threadIdx.x * VPT; offset < (page_size * num_of_layer); offset += TPB * VPT) {
      const int64_t src_layer_offset = (offset / page_size) * layer_stride;
      const int64_t dst_layer_offset = (offset / page_size) * num_of_page * page_size;
      const int64_t page_offset = offset % page_size;

      vec_copy<sizeof(int32_t) * VPT>(src + src_page_offset + src_layer_offset + page_offset, // Read from KV cache
                                      dst + dst_page_offset + dst_layer_offset + page_offset  // Store in cache block
      );
    }
  }
}

/**
 * @brief Host function for gathering cache data with 16-byte alignment.
 *
 * @tparam BlockSize Block size in bytes.
 * @tparam PageSize Page size in bytes.
 * @tparam Number of blocks launched in the kernel
 *
 * @param kvcache Pointer to the KV cache data in GPU memory.
 * @param block Output buffer in GPU memory.
 * @param kv_page_indexer Array of page indices.
 * @param num_of_page Number of pages to process.
 * @param num_of_layer Number of layers.
 * @param layer_stride Stride between layers in bytes.
 * @param stream CUDA stream for asynchronous execution.
 */
void cache_gather_align16(const char *kvcache, char *block, const int32_t *kv_page_indexer, const int64_t num_of_page,
                          const int64_t num_of_layer, const int64_t page_size, const int64_t layer_stride,
                          const int64_t page_stride, const int64_t block_size, hipStream_t stream) {
  constexpr int64_t LaunchBlockCount = ACS_KernelBlocks;

  if (page_size % 16 != 0) {
    throw std::runtime_error("Page size must be aligned to 16 bytes.");
  }
  if (block_size % 16 != 0) {
    throw std::runtime_error("Block size must be aligned to 16 bytes.");
  }
  if (block_size < page_size) {
    throw std::runtime_error("Block size must be greater than or equal to page size.");
  }
  if (page_size % sizeof(int32_t) != 0 || layer_stride % sizeof(int32_t) != 0) {
    throw std::runtime_error("Page size and layer stride must be multiples of sizeof(int32_t).");
  }
  if (page_stride % sizeof(int32_t) != 0) {
    throw std::runtime_error("Page stride must be a multiple of sizeof(int32_t).");
  }
  if (num_of_page * page_size > block_size) {
    throw std::runtime_error("Too many pages: total memory exceeds block size.");
  }

  const int64_t page_elems = page_size / sizeof(int32_t);
  const int64_t layer_stride_elems = layer_stride / sizeof(int32_t);
  const int64_t page_stride_elems = page_stride / sizeof(int32_t);

  if (page_size >= 2048) {
    constexpr int64_t TPB = 1024;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }

  switch (page_size) {
  case 64: { // Default small-page configuration
    constexpr int64_t TPB = 128;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  case 128: {
    constexpr int64_t TPB = 256;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  case 256: {
    constexpr int64_t TPB = 256;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  case 512: {
    constexpr int64_t TPB = 512;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  case 1024: {
    constexpr int64_t TPB = 512;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  case 1152: { // Legacy DeepSeek V3/R1 models
    constexpr int64_t TPB = 576;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  default: {
    constexpr int64_t TPB = 1024;
    constexpr int64_t VPT = 4;
    device_cache_gather_align16<TPB, VPT><<<LaunchBlockCount, TPB, 0, stream>>>(
        reinterpret_cast<const int32_t *>(kvcache), reinterpret_cast<int32_t *>(block), kv_page_indexer, num_of_page,
        num_of_layer, page_elems, layer_stride_elems, page_stride_elems);
    return;
  }
  }
}

} // namespace atex::ops