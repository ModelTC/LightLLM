#include "hip/hip_runtime.h"
/**
 * This is a standalone test for custom allgather.
 * To compile, make sure you have MPI and NCCL installed in your system.
 * export MPI_HOME=xxx
 * nvcc -O2 -arch=native -std=c++17 custom_all_gather_test.cu -o
 * custom_all_gather_test -lnccl -I${MPI_HOME} -lmpi
 *
 * Warning: this C++ test is not designed to be very readable and was used
 * during the rapid prototyping process.
 *
 * To run:
 * mpirun --allow-run-as-root -np 8 ./custom_all_gather_test
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#include <limits>
#include <vector>

#include "hip/hip_runtime_api.h"
#include "custom_all_gather.cuh"
#include "mpi.h"
#include "nccl.h"

#define MPICHECK(cmd)                                                  \
  do {                                                                 \
    int e = cmd;                                                       \
    if (e != MPI_SUCCESS) {                                            \
      printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e); \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

#define NCCLCHECK(cmd)                                              \
  do {                                                              \
    ncclResult_t r = cmd;                                           \
    if (r != ncclSuccess) {                                         \
      printf("Failed, NCCL error %s:%d '%s'\n", __FILE__, __LINE__, \
             ncclGetErrorString(r));                                \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

__global__ void dummy_kernel() {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  for (int i = 0; i < 100; i++) __nanosleep(1000000);  // 100ms
#else
  for (int i = 0; i < 100; i++) {
    long long int start = clock64();
    while (clock64() - start < 150000000);  // approximately 98.4ms on P40
  }
#endif
}

template <typename T>
__global__ void convert_data(const T* data1, const T* data2, double* fdata1,
                             double* fdata2, int size) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    fdata1[idx] = data1[idx];
    fdata2[idx] = data2[idx];
  }
}

__global__ void init_rand(hiprandState_t* state, int size, int nRanks) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    for (int i = 0; i < nRanks; i++) {
      hiprand_init(i + 1, idx, 0, &state[idx * nRanks + i]);
    }
  }
}

template <typename T>
void run(int myRank, int nRanks, ncclComm_t& comm, int threads, int block_limit,
         int data_size, bool performance_test) {
  T* result;
  T* result_nccl;
  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUDACHECK(hipMalloc(&result, nRanks * data_size * sizeof(T)));
  CUDACHECK(hipMemset(result, 0, nRanks * data_size * sizeof(T)));

  CUDACHECK(hipMalloc(&result_nccl, nRanks * data_size * sizeof(T)));
  CUDACHECK(hipMemset(result_nccl, 0, nRanks * data_size * sizeof(T)));

  hipIpcMemHandle_t self_data_handle;
  hipIpcMemHandle_t data_handles[8];
  vllm::Signal* buffer;
  T* self_data_copy;
  /**
   * Allocate IPC buffer
   *
   * The first section is a temporary buffer for storing intermediate allgather
   * results, if a particular algorithm requires it. The second section is for
   * the input to the allgather. The actual API takes the input pointer as an
   * argument (that is, they can and usually should be allocated separately).
   * But since the input pointers and the temporary buffer all require IPC
   * registration, they are allocated and registered together in the test for
   * convenience.
   */
  CUDACHECK(
      hipMalloc(&buffer, data_size * sizeof(T) + sizeof(vllm::Signal)));
  CUDACHECK(
      hipMemset(buffer, 0, data_size * sizeof(T) + sizeof(vllm::Signal)));
  CUDACHECK(hipMalloc(&self_data_copy, data_size * sizeof(T)));
  CUDACHECK(hipIpcGetMemHandle(&self_data_handle, buffer));

  MPICHECK(MPI_Allgather(&self_data_handle, sizeof(hipIpcMemHandle_t),
                         MPI_BYTE, data_handles, sizeof(hipIpcMemHandle_t),
                         MPI_BYTE, MPI_COMM_WORLD));

  void* rank_data;
  size_t rank_data_sz = 16 * 1024 * 1024;
  CUDACHECK(hipMalloc(&rank_data, rank_data_sz));
  vllm::Signal* ipc_ptrs[8];
  for (int i = 0; i < nRanks; i++) {
    if (i == myRank)
      ipc_ptrs[i] = buffer;
    else
      CUDACHECK(hipIpcOpenMemHandle((void**)&ipc_ptrs[i], data_handles[i],
                                     hipIpcMemLazyEnablePeerAccess));
  }
  vllm::CustomAllgather fa(ipc_ptrs, rank_data, rank_data_sz, myRank, nRanks);
  auto* self_data =
      reinterpret_cast<T*>(reinterpret_cast<char*>(buffer) +
                           sizeof(vllm::Signal));
  // hack buffer registration
  {
    void* data[8];
    for (int i = 0; i < nRanks; i++) {
      data[i] =
          ((char*)ipc_ptrs[i]) + sizeof(vllm::Signal);
    }
    fa.register_buffer(data);
  }

  double* ground_truth;
  CUDACHECK(hipHostMalloc(&ground_truth, nRanks * data_size * sizeof(double)));
  hiprandState_t* states;
  CUDACHECK(hipMalloc(&states, sizeof(hiprandState_t) * nRanks * data_size));
  init_rand<<<108, 1024, 0, stream>>>(states, data_size, nRanks);
  CUDACHECK(hipMemcpyAsync(self_data_copy, self_data, data_size * sizeof(T),
                            hipMemcpyDeviceToDevice, stream));
  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  ncclDataType_t ncclDtype;
  if (std::is_same<T, half>::value) {
    ncclDtype = ncclFloat16;
  } else if (std::is_same<T, hip_bfloat16>::value) {
    ncclDtype = ncclBfloat16;
  } else {
    ncclDtype = ncclFloat;
  }
  double *nccl_result, *my_result;
  CUDACHECK(hipHostMalloc(&nccl_result, nRanks * data_size * sizeof(double)));
  CUDACHECK(hipHostMalloc(&my_result, nRanks * data_size * sizeof(double)));
  if (performance_test) {
    dummy_kernel<<<1, 1, 0, stream>>>();
    constexpr int warmup_iters = 5;
    constexpr int num_iters = 100;
    // warmup
    for (int i = 0; i < warmup_iters; i++) {
      NCCLCHECK(ncclAllGather(self_data_copy, result_nccl, data_size, ncclDtype,
                              comm, stream));
    }
    CUDACHECK(hipEventRecord(start, stream));
    for (int i = 0; i < num_iters; i++) {
      NCCLCHECK(ncclAllGather(self_data_copy, result_nccl, data_size, ncclDtype,
                              comm, stream));
    }
    CUDACHECK(hipEventRecord(stop, stream));
    CUDACHECK(hipStreamSynchronize(stream));
    float allgather_ms = 0;
    hipEventElapsedTime(&allgather_ms, start, stop);

    dummy_kernel<<<1, 1, 0, stream>>>();
    // warm up
    for (int i = 0; i < warmup_iters; i++) {
      fa.allgather<T>(stream, self_data, result, data_size, threads,
                      block_limit);
    }
    CUDACHECK(hipEventRecord(start, stream));
    for (int i = 0; i < num_iters; i++) {
      fa.allgather<T>(stream, self_data, result, data_size, threads,
                      block_limit);
    }
    CUDACHECK(hipEventRecord(stop, stream));
    CUDACHECK(hipStreamSynchronize(stream));

    float duration_ms = 0;
    hipEventElapsedTime(&duration_ms, start, stop);
    if (myRank == 0)
      printf(
          "Rank %d done, nGPUs:%d, sz (kb): %d, %d, %d, my time:%.2fus, nccl "
          "time:%.2fus\n",
          myRank, nRanks, data_size * sizeof(T) / 1024, threads, block_limit,
          duration_ms * 1e3 / num_iters, allgather_ms * 1e3 / num_iters);

    // And wait for all the queued up work to complete
    CUDACHECK(hipStreamSynchronize(stream));

    NCCLCHECK(ncclAllGather(self_data_copy, result_nccl, data_size, ncclDtype,
                            comm, stream));

    convert_data<T><<<108, 1024, 0, stream>>>(result_nccl, result, nccl_result,
                                              my_result, nRanks * data_size);
    CUDACHECK(hipStreamSynchronize(stream));

    for (unsigned long j = 0; j < data_size; j++) {
      auto diff = abs(nccl_result[j] - my_result[j]);
      if (diff >= 4e-2) {
        printf("Rank %d: Verification mismatch at %lld: %f != (my) %f, gt=%f\n",
               myRank, j, nccl_result[j], my_result[j], ground_truth[j]);
        break;
      }
    }
    long double avg_diffs = 0.0;
    for (int j = 0; j < nRanks * data_size; j++) {
      avg_diffs += abs(nccl_result[j] - my_result[j]);
    }
    if (myRank == 0)
      std::cout << "average abs diffs: nccl: " << avg_diffs / data_size << std::endl;
      
  } else {
    for (int i = 0; i < 1; i++) {
      fa.allgather<T>(stream, self_data, result, data_size, threads,
                      block_limit);
      CUDACHECK(hipStreamSynchronize(stream));
      NCCLCHECK(ncclAllGather(self_data_copy, result_nccl, data_size, ncclDtype,
                              comm, stream));
      convert_data<T><<<108, 1024, 0, stream>>>(
          result_nccl, result, nccl_result, my_result, data_size * nRanks);
      CUDACHECK(hipStreamSynchronize(stream));

      for (unsigned long j = 0; j < data_size * nRanks; j++) {
        auto diff = abs(nccl_result[j] - my_result[j]);
        if (diff >= 4e-2) {
          printf(
              "Rank %d: Verification mismatch at %lld: %f != (my) %f, gt=%f\n",
              myRank, j, nccl_result[j], my_result[j], ground_truth[j]);
          break;
        }
      }
    }
    if (myRank == 0)
      printf("Test passed: nGPUs:%d, sz (kb): %d, %d, %d\n", nRanks,
             data_size * sizeof(T) / 1024, threads, block_limit);
    
    long double avg_diffs = 0.0;
    for (int j = 0; j < data_size * nRanks; j++) {
      avg_diffs += abs(nccl_result[j] - my_result[j]);
    }
    if (myRank == 0)
      std::cout << "average abs diffs: nccl: " << avg_diffs / data_size << std::endl;
  }

  CUDACHECK(hipFree(result));
  CUDACHECK(hipFree(self_data_copy));
  CUDACHECK(hipFree(rank_data));
  CUDACHECK(hipFree(buffer));
  CUDACHECK(hipFree(states));
  CUDACHECK(hipFree(result_nccl));
  CUDACHECK(hipHostFree(ground_truth));
  CUDACHECK(hipHostFree(nccl_result));
  CUDACHECK(hipHostFree(my_result));
  CUDACHECK(hipStreamDestroy(stream));
}

int main(int argc, char** argv) {
  int nRanks, myRank;
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
  CUDACHECK(hipSetDevice(myRank));
  ncclUniqueId id;
  ncclComm_t comm;
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast(static_cast<void*>(&id), sizeof(id), MPI_BYTE, 0,
                     MPI_COMM_WORLD));
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  bool performance_test = true;
  hipProfilerStart();
  // Uncomment to scan through different block size configs.
  // for (int threads : {256, 512, 1024}) {
  //   for (int block_limit = 16; block_limit < 112; block_limit += 4) {
  //     run<half>(myRank, nRanks, comm, threads, block_limit, 1024 * 1024,
  //     performance_test);
  //   }
  // }
  // Scan through different sizes to test performance.
  for (int sz = 512; sz <= (8 << 20); sz *= 2) {
    run<half>(myRank, nRanks, comm, 512, 36, (sz + 8 * 47), performance_test);
  }

  hipProfilerStop();
  MPICHECK(MPI_Finalize());
  return EXIT_SUCCESS;
}
